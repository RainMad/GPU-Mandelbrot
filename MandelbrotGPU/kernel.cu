#include "hip/hip_runtime.h"
#include "./kernel.h"

#include <hip/hip_complex.h>
#include <cstddef>



// per reference ! --> immer per value
// es wird auf einen anderen speicher gegriffen -> es kracht
// es werden immer ein vielfaches von 32 an Thread gestartet -> bei 33 Zeichen werden 64 Threads gestartet
// deswegen wir auch size mit�bergeben

// thread nummer ist relative zum block
// block nummer ist relativ zur grafikkarte
// daraus muss ide absolute threadnummer berechnet werden

__constant__ pfc::BGR_4_t static lookUp[128] = {
{0, 0, 0},
	{66, 30, 15},
	{25, 7, 26},
	{9, 1, 47},
	{4, 4, 73},
	{0, 7, 100},
	{12, 44, 138},
	{57, 125, 209},
	{134, 181, 229},
	{211, 236, 248},
	{241, 233, 191},
	{248, 201, 95},
	{255, 170, 0},
	{204, 128, 0},
	{153, 87, 0},
	{116, 62, 3},
	{126, 72, 13},
	{136, 82, 23},
	{146, 92, 33},
	{156, 102, 43},
	{166, 112, 53},
	{176, 122, 63},
	{186, 132, 73},
	{196, 142, 83},
	{206, 152, 93},
	{216, 162, 103},
	{217, 162, 103},
	{218, 163, 104},
	{219, 164, 105},
	{220, 165, 106},
	{221, 166, 107},
	{222, 167, 108},
	{223, 168, 109},
	{224, 169, 110},
	{225, 170, 111},
	{226, 171, 112},
	{227, 172, 113},
	{228, 173, 114},
	{229, 174, 115},
	{230, 175, 116},
	{231, 176, 117},
	{232, 177, 118},
	{233, 178, 119},
	{234, 179, 120},
	{235, 180, 121},
	{236, 181, 122},
	{237, 182, 123},
	{238, 183, 124},
	{239, 184, 125},
	{240, 185, 126},
	{241, 186, 126},
	{242, 187, 127},
	{243, 188, 128},
	{244, 189, 129},
	{245, 190, 130},
	{246, 191, 131},
	{247, 192, 132},
	{248, 193, 133},
	{249, 194, 134},
	{250, 195, 135},
	{251, 196, 136},
	{252, 197, 137},
	{253, 198, 138},
	{254, 199, 139},
	{255, 20, 140},
	{66, 30, 15},
	{25, 7, 26},
	{9, 1, 47},
	{4, 4, 73},
	{0, 7, 100},
	{12, 44, 138},
	{57, 125, 209},
	{134, 181, 229},
	{211, 236, 248},
	{241, 233, 191},
	{248, 201, 95},
	{255, 170, 0},
	{204, 128, 0},
	{153, 87, 0},
	{116, 62, 3},
	{126, 72, 13},
	{136, 82, 23},
	{146, 92, 33},
	{156, 102, 43},
	{166, 112, 53},
	{176, 122, 63},
	{186, 132, 73},
	{196, 142, 83},
	{206, 152, 93},
	{216, 162, 103},
	{217, 162, 103},
	{218, 163, 104},
	{219, 164, 105},
	{220, 165, 106},
	{221, 166, 107},
	{222, 167, 108},
	{223, 168, 109},
	{224, 169, 110},
	{225, 170, 111},
	{226, 171, 112},
	{227, 172, 113},
	{228, 173, 114},
	{229, 174, 115},
	{230, 175, 116},
	{231, 176, 117},
	{232, 177, 118},
	{233, 178, 119},
	{234, 179, 120},
	{235, 180, 121},
	{236, 181, 122},
	{237, 182, 123},
	{238, 183, 124},
	{239, 184, 125},
	{240, 185, 126},
	{241, 186, 126},
	{242, 187, 127},
	{243, 188, 128},
	{244, 189, 129},
	{245, 190, 130},
	{246, 191, 131},
	{247, 192, 132},
	{248, 193, 133},
	{249, 194, 134},
	{250, 195, 135},
	{251, 196, 136},
	{252, 197, 137},
	{253, 198, 138},
};


__device__ double pow(double x, double y);

// divergenten code vermeiden! -> eine der gr��ten Bremsen
__global__ void kernel(pfc::BGR_4_t * const p_dst, 
	std::size_t const size_x, 
	double imag_max, 
	double imag_min,
	double real_max,
	double real_min,
	int const threshold, 
	int const iteration, 
	int const bmp_width, 
	int const bmp_height, 
	int const amount_of_images, 
	double const point_real,
	double const point_imag, 
	double const zoom_factor,
	int const image_number) {
	// blockDim Anzahl der Threads pro block
	auto const t{ blockIdx.x * blockDim.x + threadIdx.x }; // -> absolute Threadnumber
	auto const u{ blockIdx.y * blockDim.y + threadIdx.y };

	if (t > bmp_width || u > bmp_height)
		return;
	int image_size = bmp_width * bmp_height;

	int x_pos = t;
	int y_pos = u;

	int image_n = t / (image_size);
	double pow_result = pow(zoom_factor, image_number + image_n);
	real_min = point_real - (point_real - real_min) * pow_result;
	real_max = point_real + (real_max - point_real) * pow_result;
	imag_max = point_imag + (imag_max - point_imag) * pow_result;
	imag_min = point_imag - (point_imag - imag_min) * pow_result;

	double x_normalize = { x_pos * 1.0 / bmp_width * (real_max - real_min) + real_min };
	double y_normalize = { y_pos * 1.0 / bmp_height * (imag_max - imag_min) + imag_min };

	pfc::complex<float> c{ x_normalize, y_normalize };
	pfc::complex<float> zi{ 0.0,0.0 };
	pfc::complex<float> zn{ 0.0,0.0 };
	int value = 0;
	for (size_t i = 0; i < iteration; i++) {
		zn = zi * zi + c;
		zi = zn;
		if (norm(zn) > threshold) {
			p_dst[u*bmp_width + t] = lookUp[i % iteration];
			break;
		}
	}

	if (norm(zn) <= threshold)
		p_dst[u*bmp_width + t] = lookUp[0];
}

// divergenten code vermeiden! -> eine der gr��ten Bremsen
__global__ void kernel1(pfc::BGR_4_t * const p_dst,
	std::size_t const size_x,
	double imag_max,
	double imag_min,
	double real_max,
	double real_min,
	int const threshold,
	int const iteration,
	int const bmp_width,
	int const bmp_height,
	int const amount_of_images,
	double const point_real,
	double const point_imag,
	double const zoom_factor,
	int const image_number) {
	//blockDim Anzahl der Threads pro block
	auto const t{ blockIdx.x * blockDim.x + threadIdx.x }; // -> absolute Threadnumber
	auto const u{ blockIdx.y * blockDim.y + threadIdx.y };

	if (t > bmp_width || u > bmp_height)
		return;
	int image_size = bmp_width * bmp_height;

	int x_pos = t;
	int y_pos = u;

	int image_n = t / (image_size);
	double pow_result = pow(zoom_factor, image_number + image_n);
	real_min = point_real - (point_real - real_min) * pow_result;
	real_max = point_real + (real_max - point_real) * pow_result;
	imag_max = point_imag + (imag_max - point_imag) * pow_result;
	imag_min = point_imag - (point_imag - imag_min) * pow_result;

	double x_normalize = { x_pos * 1.0 / bmp_width * (real_max - real_min) + real_min };
	double y_normalize = { y_pos * 1.0 / bmp_height * (imag_max - imag_min) + imag_min };


	double p = sqrt((x_normalize - 0.25)*(x_normalize - 0.25) + y_normalize * y_normalize);
	if (x_normalize <= (p - 2*p*p + 0.25) || ((x_normalize + 1)*(x_normalize + 1) + y_normalize <= 1.0/16) <= 0.25*y_normalize*y_normalize) {
		p_dst[u*bmp_width + t] = lookUp[0];
		return;
	}

	pfc::complex<float> c{ x_normalize, y_normalize };
	pfc::complex<float> zi{ 0.0,0.0 };
	pfc::complex<float> zn{ 0.0,0.0 };
	int value = 0;
	for (size_t i = 0; i < iteration; i++) {
		zn = zi * zi + c;
		zi = zn;
		if (norm(zn) > threshold) {
			p_dst[u*bmp_width + t] = lookUp[i % iteration];
			break;
			return;
		}
	}

	if (norm(zn) <= threshold)
		p_dst[u*bmp_width + t] = lookUp[0];
}

hipError_t call_kernel(
	dim3 const big, 
	dim3 const tib, 
	pfc::BGR_4_t * p_dst, 
	std::size_t const size_x, 
	double imag_max,
	double imag_min,
	double real_max,
	double real_min,
	int const threshold, 
	int const iteration, 
	int const bmp_width, 
	int const bmp_height, 
	int const amount_of_images, 
	double const point_real,
	double const point_imag, 
	double const zoom_factor,
	int const image_number) {
	// blocks in grid
	// threads in block
	// 3 kernel a 512 threads

	//kernel << <big, tib >> > (p_dst, size_x, imag_max, imag_min, real_max, real_min, threshold, iteration, bmp_width, bmp_height, amount_of_images, point_real, point_imag, zoom_factor, image_number);
	return hipGetLastError();
}

hipError_t call_kernel_1(
	dim3 const big,
	dim3 const tib,
	pfc::BGR_4_t * p_dst,
	std::size_t const size_x,
	double imag_max,
	double imag_min,
	double real_max,
	double real_min,
	int const threshold,
	int const iteration,
	int const bmp_width,
	int const bmp_height,
	int const amount_of_images,
	double const point_real,
	double const point_imag,
	double const zoom_factor,
	int const image_number) {
	// blocks in grid
	// threads in block
	// 3 kernel a 512 threads

	kernel1 << <big, tib >> > (p_dst, size_x, imag_max, imag_min, real_max, real_min, threshold, iteration, bmp_width, bmp_height, amount_of_images, point_real, point_imag, zoom_factor, image_number);
	return hipGetLastError();
}
